#include "hip/hip_runtime.h"
#include "../../../include/GPU/CUDA/cudaChangeDetection.cuh"
#include "../../../include/GPU/CUDA/changeDetectionKernel.cuh"
#define THREADS_PER_BLOCK 1024

// Variable Declarations
int gpuChoice = -1;

void printCUDADeviceProperties(void)
{
	// Code
	cout << endl << "Detected Nvidia GPU ... Using CUDA ...";
	cout << endl << "--------------------------------------------------------------------------------------------------" << endl;
	cout << endl << "CUDA INFORMATION : " << endl;
	cout << endl << "**************************************************************************************************";
	
	hipError_t retCudaRt;
	int devCount;

	retCudaRt = hipGetDeviceCount(&devCount);

	if (retCudaRt != hipSuccess)
	{
		cout << endl << "CUDA Runtime API Error - hipGetDeviceCount() Failed Due To " << hipGetErrorString(retCudaRt) << endl;
	}
	else if (devCount == 0)
	{
		cout << endl << "No CUDA Supported Devices Found On This System ... Exiting !!!" << endl;
		return;
	}
	else
	{
		for (int i = 0; i < devCount; i++)
		{
			hipDeviceProp_t devProp;
			int driverVersion = 0, runtimeVersion = 0;

			retCudaRt = hipGetDeviceProperties(&devProp, i);
			if (retCudaRt != hipSuccess)
			{
				cout << endl << " " << hipGetErrorString(retCudaRt) << "in" << __FILE__ << "at line " << __LINE__ << endl;
				return;
			}

			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);

			cout << endl << "GPU Device Number			: " << i;
			cout << endl << "GPU Device Name				: " << devProp.name;
			cout << endl << "GPU Device Memory			: " << (ceil((float)devProp.totalGlobalMem / 1048576.0f) / 1024.0f) << " GB";
			cout << endl << "GPU Device Number Of SMProcessors	: " << devProp.multiProcessorCount;
		}

		// GPU Selection
		if (devCount > 1)
		{
			cout << endl << "You have more than 1 CUDA GPU Devices ... Please select 1 of them";
			cout << endl << "Enter GPU Device Number : ";
			cin >> gpuChoice;

			// Set CUDA GPU Device
			hipSetDevice(gpuChoice);
		}
		else
		{
			// Set CUDA GPU Device
			hipSetDevice(0);
		}

		cout << endl << "**************************************************************************************************";
		cout << endl << "--------------------------------------------------------------------------------------------------" << endl;
	}
}

void convertBitmapToPixelArr(uint8_t *bitmap, Pixel *pixelArr, size_t size)
{
	for(int i=0 ; i<size ; i++, bitmap+=3)
	{
		pixelArr[i].blue = bitmap[0];
		pixelArr[i].green = bitmap[1];
		pixelArr[i].red = bitmap[2];
	}
}


void convertPixelArrToBitmap(Pixel *pixelArr, uint8_t *bitmap, size_t size){
	for(int i=0 ; i<size ; i++, bitmap+=3)
	{
		bitmap[0] = pixelArr[i].blue;
		bitmap[1] = pixelArr[i].green;
		bitmap[2] = pixelArr[i].red;
	}
}


void runOnGPU(ImageData *oldImage, ImageData *newImage, int threshold, uint8_t *detectedChanges)
{
	Pixel *h_oldImagePixArr, *h_newImagePixArr, *h_highlightedChangesPixArr;
	Pixel *d_oldImagePixArr, *d_newImagePixArr, *d_highlightedChangesPixArr;
	size_t size = (oldImage->height * oldImage->pitch)/3;
	uint8_t *bitmapPtrCpy;

	h_oldImagePixArr = (Pixel*)malloc(size * sizeof(Pixel));
	h_newImagePixArr = (Pixel*)malloc(size * sizeof(Pixel));
	h_highlightedChangesPixArr = (Pixel*)malloc(size * sizeof(Pixel));

	bitmapPtrCpy = oldImage->bitmap;
	convertBitmapToPixelArr(bitmapPtrCpy, h_oldImagePixArr, size);
	bitmapPtrCpy = newImage->bitmap;
	convertBitmapToPixelArr(bitmapPtrCpy, h_newImagePixArr, size);

	hipMalloc(&d_oldImagePixArr, size * sizeof(Pixel));
	hipMalloc(&d_newImagePixArr, size * sizeof(Pixel));
	hipMalloc(&d_highlightedChangesPixArr, size * sizeof(Pixel));

	// printCUDADeviceProperties();

	hipMemcpy(d_oldImagePixArr, h_oldImagePixArr, size * sizeof(Pixel), hipMemcpyHostToDevice);
	hipMemcpy(d_newImagePixArr, h_newImagePixArr, size * sizeof(Pixel), hipMemcpyHostToDevice);

	dim3 blocks((size + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);
	
	auto start = std::chrono::high_resolution_clock::now();
	detectChanges<<<blocks, THREADS_PER_BLOCK>>>(d_oldImagePixArr, d_newImagePixArr, d_highlightedChangesPixArr, threshold, size);
	auto stop = std::chrono::high_resolution_clock::now();

	auto GPU_Duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	cout << "GPU Duration = " << GPU_Duration.count() << endl;

	hipMemcpy(h_highlightedChangesPixArr, d_highlightedChangesPixArr, size * sizeof(Pixel), hipMemcpyDeviceToHost);

	bitmapPtrCpy = detectedChanges;
	convertPixelArrToBitmap(h_highlightedChangesPixArr, bitmapPtrCpy, size);

	free(h_oldImagePixArr);
	free(h_newImagePixArr);
	free(h_highlightedChangesPixArr);
	hipFree(d_highlightedChangesPixArr);
	hipFree(d_oldImagePixArr);
	hipFree(d_newImagePixArr);
}


void cleanup(void)
{
	cout << endl << "Placeholder Cleanup Message" << endl;
}
