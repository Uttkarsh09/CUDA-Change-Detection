#include "headers.hpp""

void listDevices(){
	int deviceCount=-1;
	int c;
	int *dev_c;
	hipDeviceProp_t deviceProperties;

	hipGetDeviceCount(&deviceCount);
	
	for(int i=0 ; i<deviceCount ; i++){
		hipGetDeviceProperties(&deviceProperties, i);
		printDeviceInformation(deviceProperties);
		printf("~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ \n");
	}
}
