#include "hip/hip_runtime.h"
#include "../../../include/GPU/CUDA/cudaChangeDetection.cuh"
#include "../../../include/GPU/CUDA/changeDetectionKernel.cuh"

#define THREADS_PER_BLOCK 1024

// Variable Declarations
Pixel *h_oldImagePixArr, *h_newImagePixArr, *h_highlightedChangePixArr;
Pixel *d_oldImagePixArr, *d_newImagePixArr, *d_highlightedChangePixArr;
int gpuChoice = -1;

void printCUDADeviceProperties(void)
{
	// Code
	cout << endl << "Detected Nvidia GPU ... Using CUDA ...";
	cout << endl << "--------------------------------------------------------------------------------------------------" << endl;
	cout << endl << "CUDA INFORMATION : " << endl;
	cout << endl << "**************************************************************************************************";
	
	hipError_t retCudaRt;
	int devCount;

	retCudaRt = hipGetDeviceCount(&devCount);

	if (retCudaRt != hipSuccess)
	{
		cout << endl << "CUDA Runtime API Error - hipGetDeviceCount() Failed Due To " << hipGetErrorString(retCudaRt) << endl;
	}
	else if (devCount == 0)
	{
		cout << endl << "No CUDA Supported Devices Found On This System ... Exiting !!!" << endl;
		return;
	}
	else
	{
		for (int i = 0; i < devCount; i++)
		{
			hipDeviceProp_t devProp;
			int driverVersion = 0, runtimeVersion = 0;

			retCudaRt = hipGetDeviceProperties(&devProp, i);
			if (retCudaRt != hipSuccess)
			{
				cout << endl << " " << hipGetErrorString(retCudaRt) << "in" << __FILE__ << "at line " << __LINE__ << endl;
				return;
			}

			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);

			cout << endl << "GPU Device Number			: " << i;
			cout << endl << "GPU Device Name				: " << devProp.name;
			cout << endl << "GPU Device Memory			: " << (ceil((float)devProp.totalGlobalMem / 1048576.0f) / 1024.0f) << " GB";
			cout << endl << "GPU Device Number Of SMProcessors	: " << devProp.multiProcessorCount;
		}

		// GPU Selection
		if (devCount > 1)
		{
			cout << endl << "You have more than 1 CUDA GPU Devices ... Please select 1 of them";
			cout << endl << "Enter GPU Device Number : ";
			cin >> gpuChoice;

			// Set CUDA GPU Device
			hipSetDevice(gpuChoice);
		}
		else
		{
			// Set CUDA GPU Device
			hipSetDevice(0);
		}

		cout << endl << "**************************************************************************************************";
		cout << endl << "--------------------------------------------------------------------------------------------------" << endl;
	}
}


void runOnGPU(ImageData *oldImage, ImageData *newImage, int threshold, uint8_t *detectedChanges)
{
	size_t size = (oldImage->height * oldImage->pitch)/3;
	float timeOnGPU = 0.0f;

	h_oldImagePixArr = (Pixel*)malloc(size * sizeof(Pixel));
	h_newImagePixArr = (Pixel*)malloc(size * sizeof(Pixel));
	h_highlightedChangePixArr = (Pixel*)malloc(size * sizeof(Pixel));

	convertBitmapToPixelArr(h_oldImagePixArr, oldImage->bitmap, size);
	convertBitmapToPixelArr(h_newImagePixArr, newImage->bitmap, size);

	hipMalloc(&d_oldImagePixArr, size * sizeof(Pixel));
	hipMalloc(&d_newImagePixArr, size * sizeof(Pixel));
	hipMalloc(&d_highlightedChangePixArr, size * sizeof(Pixel));

	printCUDADeviceProperties();

	hipMemcpy(d_oldImagePixArr, h_oldImagePixArr, size * sizeof(Pixel), hipMemcpyHostToDevice);
	hipMemcpy(d_newImagePixArr, h_newImagePixArr, size * sizeof(Pixel), hipMemcpyHostToDevice);

	dim3 blocks((size + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK);
	
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	detectChanges<<<blocks, THREADS_PER_BLOCK>>>(d_oldImagePixArr, d_newImagePixArr, d_highlightedChangePixArr, threshold, size);
	
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
 
	hipMemcpy(h_highlightedChangePixArr, d_highlightedChangePixArr, size * sizeof(Pixel), hipMemcpyDeviceToHost);

	cout << endl << "Time Taken on GPU : " << timeOnGPU << " ms" << endl;

	convertPixelArrToBitmap(detectedChanges, h_highlightedChangePixArr, size, false);

	cleanup();
}

void cleanup(void)
{
	if (d_highlightedChangePixArr)
	{
		hipFree(d_highlightedChangePixArr);
		d_highlightedChangePixArr = NULL;
	}

	if (d_newImagePixArr)
	{
		hipFree(d_newImagePixArr);
		d_newImagePixArr = NULL;
	}

	if (d_oldImagePixArr)
	{
		hipFree(d_oldImagePixArr);
		d_oldImagePixArr = NULL;
	}

	if (h_highlightedChangePixArr)
	{
		free(h_highlightedChangePixArr);
		h_highlightedChangePixArr = NULL;
	}

	if (h_newImagePixArr)
	{
		free(h_newImagePixArr);
		h_newImagePixArr = NULL;
	}

	if (h_oldImagePixArr)
	{
		free(h_oldImagePixArr);
		h_oldImagePixArr = NULL;
	}
}
